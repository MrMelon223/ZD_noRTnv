#include "hip/hip_runtime.h"
	// ZDcamera.cpp
#include "../include/ZDcamera.h"

float controller_look_sensitivity = 0.5f;

ZDcamera::ZDcamera(int_t width, int_t height, float fov, vec3_t position, vec3_t direction) {

	this->width = width;
	this->height = height;

	this->position = position;
	this->direction = glm::normalize(direction);

	this->rotation.x = atan2f(this->direction.y, this->direction.x);
	this->rotation.y = asinf(-this->direction.z);
	this->rotation.z = 0.0f;

	this->hori_fov = fov;
}

d_ZDcamera* ZDcamera::to_gpu() {
	d_ZDcamera cam{ this->position, this->direction, this->rotation, this->hori_fov, nullptr };

	cuda_check(hipMalloc((void**)&cam.vertex_samples, sizeof(d_ZDvertex_sample) * this->width * this->height));

	d_ZDcamera* d_cam;
	cuda_check(hipMalloc((void**)&d_cam, sizeof(d_ZDcamera)));
	cuda_check(hipMemcpy(d_cam, &cam, sizeof(d_ZDcamera), hipMemcpyHostToDevice));

	return d_cam;
}

d_ZDvertex_sample* from_gpu(d_ZDcamera* cam) {
	d_ZDcamera* d_cam = new d_ZDcamera{};

	cuda_check(hipMemcpy(d_cam, cam, sizeof(d_ZDcamera), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	return d_cam->vertex_samples;
}

void ZDcamera::update_direction(float x, float y) {
	//printf("X,Y input mouse coord = {%.2f, %.2f}\n", rot.x, rot.y);
	float normalized_coord_x = ((x - (static_cast<float>(this->width) * 0.5f)) / static_cast<float>(this->width));
	float normalized_coord_y = ((y - (static_cast<float>(this->height) * 0.5f)) / static_cast<float>(this->height));
	//printf("X,Y normalized input mouse coord = {%.2f, %.2f}\n", normalized_coord_x, normalized_coord_y);

	float aspect_ratio = static_cast<float>(this->width) / static_cast<float>(this->height);

	float fov_hori_rad = this->hori_fov;
	float fov_vert_rad = this->hori_fov * (static_cast<float>(this->height) / static_cast<float>(this->width));
	float half_fov_hori_rad = fov_hori_rad * 0.5f;
	float half_fov_vert_rad = fov_vert_rad * 0.5f;

	float view_x = normalized_coord_x * half_fov_hori_rad * aspect_ratio;
	float view_y = normalized_coord_y * half_fov_vert_rad;

	this->rotation.y += view_x * 0.01f; //* (static_cast<float>(this->dims.x) / this->dims.y);
	this->rotation.x += view_y * 0.01f;
	this->rotation.z = 0.0f;

	if (this->rotation.x > PI * 0.5f) {
		this->rotation.x = PI * 0.5f;
	}
	if (this->rotation.x < -PI * 0.5f) {
		this->rotation.x = -PI * 0.5f;
	}
	this->rotation.x = fmod(this->rotation.x, 2.0f * PI);
	this->rotation.y = fmod(this->rotation.y, 2.0f * PI);

	float yaw = this->rotation.x * (PI / 180.0f),
		pitch = this->rotation.y * (PI / 180.0f);

	this->direction.x = cosf(yaw) * cosf(pitch);
	this->direction.y = sinf(pitch);
	this->direction.z = sinf(yaw) * cosf(pitch);

	this->direction = glm::normalize(this->direction);
}

void ZDcamera::update_direction() {

	this->rotation.x = fmod(this->rotation.x, 2.0f * PI);
	this->rotation.y = fmod(this->rotation.y, 2.0f * PI);

	if (this->rotation.x > PI * 0.5f) {
		this->rotation.x = PI * 0.5f;
	}
	if (this->rotation.x < -PI * 0.5f) {
		this->rotation.x = -PI * 0.5f;
	}

	float yaw = this->rotation.x,
		pitch = this->rotation.y;

	this->direction.x = cosf(yaw) * cosf(pitch);
	this->direction.y = sinf(pitch);
	this->direction.z = sinf(yaw) * cosf(pitch);

	this->direction = glm::normalize(this->direction);
}

void ZDcamera::forward(float t) {
	this->position = this->position + (glm::normalize(this->direction) * t * 0.5f);

	if (this->position.x < 0.0f) {
		this->position.x = 0.0f;
	}
	if (this->position.y < 0.0f) {
		this->position.y = 0.0f;
	}
	if (this->position.z < 0.0f) {
		this->position.z = 0.0f;
	}
}

void ZDcamera::backward(float t) {
	this->position = this->position - glm::normalize(this->direction) * t * 0.5f;
}
void ZDcamera::right(float t) {
	this->position = this->position - glm::cross(this->direction, vec3_t(0.0f, 1.0f, 0.0f)) * t * 0.5f;
}
void ZDcamera::left(float t) {
	this->position = this->position + glm::cross(this->direction, vec3_t(0.0f, 1.0f, 0.0f)) * t * 0.5f;
}

void ZDcamera::turn_right(float t) {
	float rot = 0.1f * controller_look_sensitivity;
	if (rot >= ZDcamera::DEADZONE) {
		this->rotation.y += rot;
		this->rotation.y = fmod(this->rotation.y, 2.0f * PI);
	}
}
void ZDcamera::turn_left(float t) {
	float rot = 0.1f * controller_look_sensitivity;
	if (rot >= ZDcamera::DEADZONE) {
		this->rotation.y -= rot;
		this->rotation.y = fmod(this->rotation.y, 2.0f * PI);
	}
}

void ZDcamera::turn_up(float t) {
	float rot = 0.1f * controller_look_sensitivity;
	if (rot >= ZDcamera::DEADZONE) {
		this->rotation.x -= rot;
		this->rotation.x = fmod(this->rotation.y, 2.0f * PI);
	}
}
void ZDcamera::turn_down(float t) {
	float rot = 0.1f * controller_look_sensitivity;
	if (rot >= ZDcamera::DEADZONE) {
		this->rotation.x += rot;
		this->rotation.x = fmod(this->rotation.x, 2.0f * PI);
	}
}

void ZDcamera::turn_right_for(float t) {
	t = t / static_cast<float>(this->width);
	if (t >= DEADZONE || t <= -DEADZONE) {
		this->rotation.y += t * 0.1f * controller_look_sensitivity;

		if (this->rotation.y < 0.0f) {
			this->rotation.y = fmod(this->rotation.y, -2.0f * PI);
		}
		else {
			this->rotation.y = fmod(this->rotation.y, 2.0f * PI);
		}

	}
}
void ZDcamera::look_up_for(float t) {
	t = t / static_cast<float>(this->height);
	if (t >= DEADZONE || t <= -DEADZONE) {
		this->rotation.x += t * 0.1f * controller_look_sensitivity;
	}
	if (this->rotation.x < 0.0f) {
		this->rotation.x = fmod(this->rotation.x, -2.0f * PI);
	}
	else {
		this->rotation.x = fmod(this->rotation.x, 2.0f * PI);
	}

	if (this->rotation.x > PI / 2.0f) {
		this->rotation.x = PI / 2.0f;
	}
	if (this->rotation.x < -PI / 2.0f) {
		this->rotation.x = -PI / 2.0f;
	}
}

void ZDcamera::debug_print() {
	std::cout << "Camera Debug:" << std::endl;
	std::cout << std::setw(15) << "Res: " << this->width << "x" << this->height << "p" << std::endl;
	std::cout << std::setw(15) << "Loc: { " << this->position.x << ", " << this->position.y << ", " << this->position.z << " }" << std::endl;
	std::cout << std::setw(15) << "Dir: { " << this->direction.x << ", " << this->direction.y << ", " << this->direction.z << " }" << std::endl;
}